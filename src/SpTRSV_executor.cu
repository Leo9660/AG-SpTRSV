#include "hip/hip_runtime.h"
#include "../include/AG-SpTRSV.h"
#include <hip/hip_runtime.h>

// In this implementation, we assume that
// diagonal elements of the matrix are explicitly stored, 
// together with off-diagonal elements in CSR format
template<typename T>
__global__ void SpTRSV_simple(int *level, node_info **info, 
            const int *csrRowPtr, const int *csrColIdx, const T* csrValue,
            const T* b, T* x, int *get_value)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int wid = tid / WARP_SIZE;
    int global_wid = bid * WARP_NUM_PER_BLOCK + wid;
    int lane_id = tid % WARP_SIZE;

    for (int schedule_i = 0; schedule_i < level[global_wid]; schedule_i++)
    {
        node_info current_info = info[global_wid][schedule_i];

        int row_st = current_info.start_row;
        int row_ed = current_info.end_row;

        SYNC_ELIM elim = current_info.elim;

        // One warp for one row
        if (elim == NO_ELIM)
        {
            #define WRITE_FENCE __threadfence()
            #define READ_FENCE __threadfence()

            #include "SpTRSV_executor_code.cu"

            #undef WRITE_FENCE
            #undef READ_FENCE
        }
        else if (elim == NO_WRITE_FENCE)
        {
            #define WRITE_FENCE
            #define READ_FENCE __threadfence()

            #include "SpTRSV_executor_code.cu"

            #undef WRITE_FENCE
            #undef READ_FENCE
        }
        else if (elim == WRITE_FENCE_BLOCK)
        {
            #define WRITE_FENCE __threadfence_block()
            #define READ_FENCE __threadfence()

            #include "SpTRSV_executor_code.cu"

            #undef WRITE_FENCE
            #undef READ_FENCE
        }
    }
}

template<typename T>
__global__ void SpTRSV_simple_no_schedule(node_info *info, int total_node,
            const int *csrRowPtr, const int *csrColIdx, const T* csrValue,
            const T* b, T* x, int *get_value)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int wid = tid / WARP_SIZE;
    int global_wid = bid * WARP_NUM_PER_BLOCK + wid;
    int lane_id = tid % WARP_SIZE;

    if (global_wid > total_node) return;

    node_info current_info = info[global_wid];

    int row_st = current_info.start_row;
    int row_ed = current_info.end_row;

    SYNC_ELIM elim = current_info.elim;

    if (elim == NO_ELIM)
    {
        #define WRITE_FENCE __threadfence()
        #define READ_FENCE __threadfence()

        #include "SpTRSV_executor_code.cu"

        #undef WRITE_FENCE
        #undef READ_FENCE
    }
    else if (elim == NO_WRITE_FENCE)
    {
        #define WRITE_FENCE
        #define READ_FENCE __threadfence()

        #include "SpTRSV_executor_code.cu"

        #undef WRITE_FENCE
        #undef READ_FENCE
    }
    else if (elim == WRITE_FENCE_BLOCK)
    {
        #define WRITE_FENCE __threadfence_block()
        #define READ_FENCE __threadfence()

        #include "SpTRSV_executor_code.cu"

        #undef WRITE_FENCE
        #undef READ_FENCE
    }
}

template <typename T>
void SpTRSV_executor(ptr_handler handler, 
            const int *csrRowPtr_d, const int *csrColIdx_d, const T* csrValue_d,
            const T* b_d, T* x_d)
{
    if (handler->sched_s != SEQUENTIAL2)
        SpTRSV_simple<T><<<BLOCK_NUM, THREAD_NUM_PER_BLOCK>>>(handler->schedule_level_d, 
        handler->schedule_info_d, csrRowPtr_d, csrColIdx_d, csrValue_d, 
        b_d, x_d, handler->get_value);
    else
    {
        int total_node = handler->graph->global_node;
        int total_warp_num = total_node;
        int total_thread_num = total_node * WARP_SIZE;
        int block_num = (total_thread_num - 1) / THREAD_NUM_PER_BLOCK + 1;

        SpTRSV_simple_no_schedule<T><<<block_num, THREAD_NUM_PER_BLOCK>>>
        (handler->no_schedule_info_d, total_node, csrRowPtr_d, csrColIdx_d, csrValue_d,
        b_d, x_d, handler->get_value);
    }
}

// In development
template<typename T>
__global__ void SpTRSV_hybrid(int *level, node_info **info, 
            const int *RowPtr, const int *ValPtr, const int *idx, const T* value,
            const T* b, T* x, int *get_value)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int wid = tid / WARP_SIZE;
    int global_wid = bid * WARP_NUM_PER_BLOCK + wid;
    int lane_id = tid % WARP_SIZE;

    for (int schedule_i = 0; schedule_i < level[global_wid]; schedule_i++)
    {
        node_info current_info = info[global_wid][schedule_i];

        int row_st = current_info.start_row;
        int row_ed = current_info.end_row;

        if (current_info.format == TELL)
        {
            int row_num = row_ed - row_st;
            int idx_st = RowPtr[row_st];
            int idx_ed = RowPtr[row_ed];
            int val_st = ValPtr[row_st];
            int val_ed = ValPtr[row_ed];

            for (int row_iter = row_st + lane_id; row_iter < row_ed; row_iter += WARP_SIZE)
            {
                T leftsum = 0;
                T rh_value = b[row_iter];

                __threadfence();

                int row_offset = row_iter - row_st;
                int val_i = val_st + row_offset;
                for (int idx_i = idx_st + row_offset; idx_i < idx_ed; idx_i += row_num)
                {
                    int dep_row = idx[idx_i];
                    while (!get_value[dep_row])
                    {
                        __threadfence();
                    }
                    leftsum += value[val_i] * x[dep_row];
                    val_i += row_num;
                }
                x[row_iter] = (rh_value - leftsum) / value[val_i];
                __threadfence();
                get_value[row_iter] = 1;
            }
        }
        else if (current_info.format == CSR)
        {
            printf("Not implemented!\n");
        }
    }
}

template <typename T>
void SpTRSV_executor_hybrid(ptr_handler handler, 
            const int *RowPtr_d, const int *ValPtr_d,
            const int *idx_d, const T* value_d,
            const T* b_d, T* x_d)
{
    SpTRSV_hybrid<T><<<BLOCK_NUM, THREAD_NUM_PER_BLOCK>>>(handler->schedule_level_d, 
    handler->schedule_info_d, RowPtr_d, ValPtr_d, idx_d, value_d, 
    b_d, x_d, handler->get_value);
}

// instance
template void SpTRSV_executor<float>(ptr_handler handler, 
            const int *csrRowPtr, const int *csrColIdx, const float* csrValue,
            const float* b, float* x);

template void SpTRSV_executor<double>(ptr_handler handler, 
            const int *csrRowPtr, const int *csrColIdx, const double* csrValue,
            const double* b, double* x);

template void SpTRSV_executor_hybrid<float>(ptr_handler handler, 
            const int *RowPtr_d, const int *ValPtr_d,
            const int *idx_d, const float* value_d,
            const float* b_d, float* x_d);

template void SpTRSV_executor_hybrid<double>(ptr_handler handler, 
            const int *RowPtr_d, const int *ValPtr_d,
            const int *idx_d, const double* value_d,
            const double* b_d, double* x_d);