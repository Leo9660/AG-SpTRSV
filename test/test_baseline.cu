#include "hip/hip_runtime.h"
#include "test.h"
#include "YYSpTRSV.h"

struct timeval tv_begin, tv_end;

float test_cusparse(int m, int nnzL, int *csrRowPtr_d, int *csrColIdx_d, 
VALUE_TYPE *csrValue_d, VALUE_TYPE *b_d, VALUE_TYPE *x_d)
{
    // cuSparse
    hipsparseHandle_t cusparse_handler;
    hipsparseStatus_t ErrorStatus;
    ErrorStatus = hipsparseCreate(&cusparse_handler);

    hipsparseMatDescr_t desc;
    ErrorStatus = hipsparseCreateMatDescr(&desc);
    hipsparseSetMatIndexBase(desc, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatFillMode(desc, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(desc, HIPSPARSE_DIAG_TYPE_NON_UNIT);

    bsrsv2Info_t cusparse_info;
    hipsparseCreateBsrsv2Info(&cusparse_info);

    int buffer_size;
#if (VALUE_SIZE == 4)
    ErrorStatus = hipsparseSbsrsv2_bufferSize(cusparse_handler,
    HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, nnzL,
    desc, csrValue_d, csrRowPtr_d, csrColIdx_d, 1, cusparse_info, &buffer_size);
#else
    ErrorStatus = hipsparseDbsrsv2_bufferSize(cusparse_handler,
    HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, nnzL,
    desc, csrValue_d, csrRowPtr_d, csrColIdx_d, 1, cusparse_info, &buffer_size);
#endif

    if (ErrorStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        printf("Error in buffersize stage!\n");
        exit(1);
    }

    void *cusparse_buffer;
    hipMalloc((void **)&cusparse_buffer, buffer_size);

#if (VALUE_SIZE == 4)
    ErrorStatus = hipsparseSbsrsv2_analysis(cusparse_handler,
    HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, nnzL,
    desc, csrValue_d, csrRowPtr_d, csrColIdx_d, 1, cusparse_info, 
    HIPSPARSE_SOLVE_POLICY_USE_LEVEL, cusparse_buffer);
#else
    ErrorStatus = hipsparseDbsrsv2_analysis(cusparse_handler,
    HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, nnzL,
    desc, csrValue_d, csrRowPtr_d, csrColIdx_d, 1, cusparse_info, 
    HIPSPARSE_SOLVE_POLICY_USE_LEVEL, cusparse_buffer);
#endif
    if (ErrorStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        printf("Error in analysis stage!\n");
        printf("%s\n", hipsparseGetErrorString(ErrorStatus));
        exit(1);
    }
    int structural_zero;
    ErrorStatus = hipsparseXbsrsv2_zeroPivot(cusparse_handler, cusparse_info, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == ErrorStatus)
        printf("L(%d,%d) is missing\n", structural_zero, structural_zero);

    VALUE_TYPE alpha = 1.0;

    float cusparse_time = 0;

    for (int i = 0; i < REPEAT_TIME; i++)
    {
        hipMemset(x_d, 0, m * sizeof(VALUE_TYPE));

        gettimeofday(&tv_begin, NULL);

#if (VALUE_SIZE == 4)
        ErrorStatus = hipsparseSbsrsv2_solve(cusparse_handler,
        HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, nnzL,
        &alpha, desc, csrValue_d, csrRowPtr_d, csrColIdx_d, 1, cusparse_info,
        b_d, x_d, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, cusparse_buffer);
#else
        ErrorStatus = hipsparseDbsrsv2_solve(cusparse_handler,
        HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, nnzL,
        &alpha, desc, csrValue_d, csrRowPtr_d, csrColIdx_d, 1, cusparse_info,
        b_d, x_d, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, cusparse_buffer);
#endif
        hipDeviceSynchronize();

        gettimeofday(&tv_end, NULL);
        
        cusparse_time += duration(tv_begin, tv_end);

        if (ErrorStatus != HIPSPARSE_STATUS_SUCCESS)
        {
            printf("Error in solve stage!\n");
            exit(1);
        }
    }

    // L has unit diagonal, so no numerical zero is reported.
    int numerical_zero;
    ErrorStatus = hipsparseXbsrsv2_zeroPivot(cusparse_handler, cusparse_info, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == ErrorStatus){
        printf("L(%d,%d) is zero\n", numerical_zero, numerical_zero);
    }

    cusparse_time /= REPEAT_TIME;

    hipFree(cusparse_buffer);

    return cusparse_time;
}

float test_yy(int m, int nnzL, int *csrRowPtr, int *csrColIdx, VALUE_TYPE *csrValue,
int *csrRowPtr_d, int *csrColIdx_d, VALUE_TYPE *csrValue_d, 
VALUE_TYPE *b_d, VALUE_TYPE *x_d, int avg_thresh)
{

    // YYSpTRSV
    int Len;
    int *warp_num=(int *)malloc((m+1)*sizeof(int));
    if (warp_num==NULL)
        printf("warp_num error\n");
    memset(warp_num, 0, sizeof(int)*(m+1));
    
    double warp_occupy=0,element_occupy=0;
    matrix_warp(m, m, nnzL, csrRowPtr, csrColIdx,
    avg_thresh, &Len, warp_num, &warp_occupy, &element_occupy);

    int *d_warp_num;
    hipMalloc((void **)&d_warp_num, Len  * sizeof(int));
    hipMemcpy(d_warp_num, warp_num, Len * sizeof(int), hipMemcpyHostToDevice);

    int num_threads = WARP_PER_BLOCK * WARP_SIZE;
    int num_blocks = ceil ((double)((Len-1)*WARP_SIZE) / (double)(num_threads));
    
    int *d_id_extractor;
    hipMalloc((void **)&d_id_extractor, sizeof(int));

    int *yy_get_value;
    hipMalloc(&yy_get_value, m * sizeof(int));
    hipMemset(yy_get_value, 0, sizeof(int) * m);
    
    float yy_time = 0;

    for (int i = 0; i < REPEAT_TIME; i++)
    {
        hipMemset(yy_get_value, 0, sizeof(int) * m);
        hipMemset(d_id_extractor, 0, sizeof(int));
        hipMemset(x_d, 0, m * sizeof(VALUE_TYPE));

        hipDeviceSynchronize();
        
        gettimeofday(&tv_begin, NULL);
        
        yySpTRSV_csr_kernel<<< num_blocks, num_threads >>> (csrRowPtr_d, csrColIdx_d, 
        csrValue_d, yy_get_value, m, nnzL, b_d, x_d, 0, d_warp_num, Len, d_id_extractor);
        hipDeviceSynchronize();
        
        gettimeofday(&tv_end, NULL);
        
        yy_time += duration(tv_begin, tv_end);
        
    }

    yy_time /= REPEAT_TIME;

    hipFree(d_warp_num);
    hipFree(d_id_extractor);
    hipFree(yy_get_value);

    return yy_time;
}

int cmp_vector(int m, const char* name1, const char* name2, VALUE_TYPE *x1, VALUE_TYPE *x2)
{
    for (int i = 0; i < m; i++)
    {
        if (fabs(x1[i] - x2[i]) > ERROR_THRESH)
        {
            printf("%s vs %s error at index %d, x1 = %.5f, x2 = %.5f!\n", 
            name1, name2, i, x1[i], x2[i]);
            return 0;
        }
    }
    printf("%s vs %s correct!\n", name1, name2);
    return 1;
}