#include "test.h"

extern float test_cusparse(int m, int nnzL, int *csrRowPtr_d, int *csrColIdx_d, 
VALUE_TYPE *csrValue_d, VALUE_TYPE *b_d, VALUE_TYPE *x_d);
extern float test_yy(int m, int nnzL, int *csrRowPtr, int *csrColIdx, VALUE_TYPE *csrValue,
int *csrRowPtr_d, int *csrColIdx_d, VALUE_TYPE *csrValue_d, 
VALUE_TYPE *b_d, VALUE_TYPE *x_d, int avg_thresh);
int cmp_vector(int m, const char* name1, const char* name2, VALUE_TYPE *x1, VALUE_TYPE *x2);

int main(int argc, char* argv[])
{
    struct timeval tv_begin, tv_end;

    int ch;
    int input_flag = 0;
    char *input_name;

    while ((ch = getopt(argc, argv, "i:")) != -1)
    {
        switch (ch)
        {
            case 'i':
                input_flag = 1;
                input_name = optarg;
                break;
        }
    }

    if (input_flag == 0)
    {
        printf("[Usage]: ./main_batch -i {input_filename}\n");
        exit(1);
    }

    // Triangular matrix L;
    int m;
    int nnzL;
    int *csrRowPtrL;
    int *csrColIdxL;
    VALUE_TYPE *csrValL;

    read_tri<VALUE_TYPE>(input_name, &m, &nnzL, &csrRowPtrL, &csrColIdxL, &csrValL);

    int layer;
    double parallelism;
    matrix_layer(m, m, nnzL, csrRowPtrL, csrColIdxL, &layer, &parallelism);

    // print matrix information
    printf("matrix information: location %s\n"
            "m %d nnz %d layer %d parallelism %.2f\n", 
            input_name, m, nnzL, layer, parallelism);

    // x & randomized b
    VALUE_TYPE *x, *b;
    x = (VALUE_TYPE*)malloc(m * sizeof(VALUE_TYPE));
    b = (VALUE_TYPE*)malloc(m * sizeof(VALUE_TYPE));
    srand(0);
    for (int i = 0; i < m; i++)
    {
        b[i] = rand() * 1.0 / RAND_MAX;
    }

    // copy matrix and vector from CPU to GPU memory
    int *csrRowPtr_d, *csrColIdx_d;
    VALUE_TYPE *csrValL_d, *b_d, *x_d;
    hipMalloc(&csrRowPtr_d, (m + 1) * sizeof(int));
    hipMemcpy(csrRowPtr_d, csrRowPtrL, (m + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&csrColIdx_d, nnzL * sizeof(int));
    hipMemcpy(csrColIdx_d, csrColIdxL, nnzL * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&csrValL_d, nnzL * sizeof(VALUE_TYPE));
    hipMemcpy(csrValL_d, csrValL, nnzL * sizeof(VALUE_TYPE), hipMemcpyHostToDevice);
    hipMalloc(&b_d, m * sizeof(VALUE_TYPE));
    hipMemcpy(b_d, b, m * sizeof(VALUE_TYPE), hipMemcpyHostToDevice);
    hipMalloc(&x_d, m * sizeof(VALUE_TYPE));
    hipMemset(x_d, 0, sizeof(VALUE_TYPE) * m);

//--------------------AG-SpTRSV-RUN--------------------//

    // strategies
    PREPROCESSING_STRATEGY ps = ROW_BLOCK;
    SCHEDULE_STRATEGY ss = SIMPLE;
    int rb = 1;

    // preprocessing and get handler
    ptr_handler handler = SpTRSV_preprocessing(m, nnzL, csrRowPtrL, csrColIdxL, ps, rb);

    sptrsv_schedule(handler, ss);

    float sptrsv_time;
    for (int i = 0; i < REPEAT_TIME; i++)
    {
        hipMemset(handler->get_value, 0, sizeof(int) * m);

        gettimeofday(&tv_begin, NULL);
        
        SpTRSV_executor<VALUE_TYPE>(handler, csrRowPtr_d, csrColIdx_d, csrValL_d, b_d, x_d);
        hipDeviceSynchronize();

        gettimeofday(&tv_end, NULL);

        sptrsv_time += duration(tv_begin, tv_end);
    }

    sptrsv_time /= REPEAT_TIME;

    hipMemcpy(x, x_d, m * sizeof(VALUE_TYPE), hipMemcpyDeviceToHost);

    SpTRSV_finalize(handler);

    delete handler;

    printf("AG solve time: %.2f us\n", sptrsv_time);

//--------------------AG-SpTRSV-END--------------------//

    // solution vector of on host memory for correctness test
    VALUE_TYPE *x_base;
    x_base = (VALUE_TYPE*)malloc(m * sizeof(VALUE_TYPE));

#if (CU_TEST == true)

    float cusparse_time = test_cusparse(m, nnzL, csrRowPtr_d, csrColIdx_d, csrValL_d, b_d, x_d);

    memset(x_base, 0, sizeof(VALUE_TYPE)*m);
    hipMemcpy(x_base, x_d, m * sizeof(VALUE_TYPE), hipMemcpyDeviceToHost);

    cmp_vector(m, "AG", "cuSPARSE", x, x_base);

    printf("cuSPARSE solve time: %.2f us\n", cusparse_time);
    printf("AG Speedup over cuSPARSE: %.2f\n", cusparse_time / sptrsv_time);

#endif

#if (YY_TEST == true)

    float yy_time = test_yy(m, nnzL, csrRowPtrL, csrColIdxL, csrValL,
    csrRowPtr_d, csrColIdx_d, csrValL_d, b_d, x_d, 10);

    memset(x_base, 0, sizeof(VALUE_TYPE)*m);
    hipMemcpy(x_base, x_d, m * sizeof(VALUE_TYPE), hipMemcpyDeviceToHost);

    cmp_vector(m, "AG", "YY", x, x_base);

    printf("YYSpTRSV solve time: %.2f us\n", yy_time);
    printf("AG Speedup over YYSpTRSV: %.2f\n", yy_time / sptrsv_time);

#endif

    // Finalize
    hipFree(csrRowPtr_d);
    hipFree(csrColIdx_d);
    hipFree(csrValL_d);
    hipFree(x_d);
    hipFree(b_d);

}
